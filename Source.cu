#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#define LOADBMP_IMPLEMENTATION
#include "ImageHandler.h"
#include "ImageModel.h"
#include <stdio.h>
#include <math.h>
#include "bmpLoader.h"
#include "helper.h"

#define FILTER_SIZE 3
enum Filter
{
	BoxBlur = 0,
	Sharpen = 1
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void convolution(float* pixelMap, int* filter, double coef, float* resultMap, int width, int height, int components) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	float f[] = { 0, -1, 0, -1, 5, -1, 0, -1, 0 };
	// double f[] = {0, 0, 0, 0, 1, 0, 0, 0, 0};
	// double f[] = {-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,};
	if (i >= width || j >= height)
		return;
	for (int z = 0; z < components; z++) {// itterate thru channels
		float sum = 0.0;
		for (int x = -(FILTER_SIZE / 2); x <= (FILTER_SIZE / 2); x++) // itterate thru filter rows
			for (int y = -(FILTER_SIZE / 2); y <= (FILTER_SIZE / 2); y++) { // itterate thru filter cols
				float pixel = pixelMap[((i + x) * width + (j + y)) * components + z];
				float ff = f[(x + 1) * FILTER_SIZE + (y + 1)];
				sum += (i + x >= width || i + x < 0 || y + j >= height || y + j < 0)
					? 0
					: ff * pixel;
			}
		resultMap[(i * width + j) * components + z] = sum;
	}
}

// double coef[2] = { 1, 1.0 };
int filters[2][3][3] = {
	{
		{1,1,1},
		{1,1,1},
		{1,1,1}
	},
	{
		{0,-1,0},
		{-1,5,-1},
		{0,-1,0}
	}
};

int main(char** argv, int argc) {
	float* d_pixelMap, * d_resultMap, * h_resultMap;
	int size;
	auto inputImage = importPPM("lena.ppm");
	exportPPM("test.ppm", inputImage);
	auto outputImage = Image_new(inputImage->width, inputImage->height, inputImage->channels);
	size = inputImage->width * inputImage->height * inputImage->channels * sizeof(float);
	//int* flatFilter = (int*)flattenArray((void**)filters[BoxBlur], 3, 3, sizeof(int));
	int flatFilter[] = { 0,-1,0,-1,5,-1,0,-1,0 };
	// float* d_pixelMap, *d_resultMap, *h_resultMap;
	int* d_filter;
	//gpuErrchk(hipMalloc((void**)&d_filter, sizeof(int) * 3 * 3));
	gpuErrchk(hipMalloc((void**)&d_pixelMap, sizeof(float) * size));
	gpuErrchk(hipMalloc((void**)&d_resultMap, sizeof(float) * size));
	//---cpy

	gpuErrchk(hipMemcpy(d_pixelMap, inputImage->data, size, hipMemcpyHostToDevice));
	// gpuErrchk(hipMemcpy(d_filter, flatFilter, sizeof(int) * 9, hipMemcpyHostToDevice));
	//DO STUFF
	/*
	Declare and allocate host and device memory. <
	Initialize host data. <
	Transfer data from the host to the device. <
	Execute one or more kernels. <
	Transfer results from the device to the host. <
	*/
	printf("%d %d\n", inputImage->width, inputImage->height);
	dim3 numberOfBlocks(512 / 32, 512 / 32);
	dim3 threadsPerBlock(32, 32);
	hipDeviceSynchronize();
	convolution << <numberOfBlocks, threadsPerBlock >> > (d_pixelMap, NULL, 0.0, d_resultMap, inputImage->width, inputImage->height, 3);

	gpuErrchk(hipPeekAtLastError());
	h_resultMap = (float*)malloc(sizeof(float) * 512 * 512 * 3);

	gpuErrchk(hipMemcpy(h_resultMap, d_resultMap, size, hipMemcpyDeviceToHost));

	// hipDeviceSynchronize();
	// loadbmp_encode_file("lena2.bmp", h_resultMap, width, height, LOADBMP_RGB);
	outputImage->data = h_resultMap;
	exportPPM("output.ppm", outputImage);

	free(h_resultMap);
	// free(flatFilter);
	// fclose(fp);
	// hipFree(d_filter);
	hipFree(d_resultMap);
	hipFree(d_pixelMap);
	return 0;
}