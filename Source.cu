#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "ImageHandler.h"
#include "ImageModel.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "helper.h"

#define BLOCK_SIZE 32 // ideal blocksize for preformance: 32 x 32 = 1024 => block core count
enum Filter
{
	BoxBlur = 0,
	Sharpen = 1
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}
typedef unsigned char byte_t;
__global__ void convolution(float* pixelMap, float* filter, float* resultMap, int width, int height, int components, const int FILTER_SIZE) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	const int filterRadius = FILTER_SIZE / 2;
	if (i >= width || j >= height)
		return;
	for (int z = 0; z < components; z++) {// itterate thru colors
		float sum = 0.0;
		for (int x = -filterRadius; x <= filterRadius; x++) // itterate thru filter rows
			for (int y = -filterRadius; y <= filterRadius; y++) // itterate thru filter cols
				sum += (i + x >= width || i + x < 0 || y + j >= height || y + j < 0)
					? 0 // edge ignore solution
					: filter[(x + 1) * FILTER_SIZE + (y + 1)] // filter x pixel[color]
						* pixelMap[((i + x) * width + (j + y)) * components + z];
		resultMap[(i * width + j) * components + z] = sum;
	}
}

int main(char** argv, int argc) {
	float* d_pixelMap, * d_resultMap, * h_resultMap, *** filters;
	char** filter_names;
	int* filter_sizes, filter_count;
	int size;
	readFilters("filters.txt",&filters,&filter_sizes,&filter_names, &filter_count);
	int pick = showMenu(filter_names, filter_count);
	const int FILTER_SIZE = filter_sizes[pick];

	auto inputImage = importPPM("lena.ppm");
	auto outputImage = Image_new(inputImage->width, inputImage->height, inputImage->channels);
	size = inputImage->width * inputImage->height * inputImage->channels;
	float* flatFilter = flatenFilter(filters[pick], FILTER_SIZE);
	float* d_filter;
	/*
	Declare and allocate host and device memory. <
	Initialize host data. <
	Transfer data from the host to the device. <
	Execute one or more kernels. <
	Transfer results from the device to the host. <
	*/
	// malloc
	gpuErrchk(hipMalloc((void**)&d_filter, sizeof(float) * FILTER_SIZE * FILTER_SIZE));
	gpuErrchk(hipMalloc((void**)&d_pixelMap, sizeof(float) * size));
	gpuErrchk(hipMalloc((void**)&d_resultMap, sizeof(float) * size));
	//---cpy
	gpuErrchk(hipMemcpy(d_pixelMap, inputImage->data, size * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_filter, flatFilter, sizeof(float) * FILTER_SIZE * FILTER_SIZE, hipMemcpyHostToDevice));
	//DO STUFF

	dim3 numberOfBlocks(ceil(inputImage->width) / BLOCK_SIZE, ceil(inputImage->height / BLOCK_SIZE));
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	// hipDeviceSynchronize();
	convolution << <numberOfBlocks, threadsPerBlock >> > (d_pixelMap, d_filter, d_resultMap, inputImage->width, inputImage->height, 3, FILTER_SIZE);

	gpuErrchk(hipPeekAtLastError());
	h_resultMap = (float*)malloc(sizeof(float) * inputImage->width * inputImage->height * inputImage->channels);

	gpuErrchk(hipMemcpy(h_resultMap, d_resultMap, size * sizeof(float), hipMemcpyDeviceToHost));

	outputImage->data = h_resultMap;
	exportPPM("output.ppm", outputImage);

	//clean up
	free(h_resultMap);
	free(flatFilter);
	hipFree(d_filter);
	hipFree(d_resultMap);
	hipFree(d_pixelMap);
	return 0;
}