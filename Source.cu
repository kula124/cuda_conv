#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#define LOADBMP_IMPLEMENTATION
#include "ImageHandler.h"
#include "ImageModel.h"
#include <stdio.h>
#include <math.h>
#include "bmpLoader.h"
#include "helper.h"

#define FILTER_SIZE 3
enum Filter
{
	BoxBlur = 0,
	Sharpen = 1
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

// FILE* fp = fopen("./wtf.txt", "w+");

typedef float byte_t;
__global__ void convolution(byte_t* pixelMap, int* filter, double coef, byte_t* resultMap, int width, int height, int channels) {
	// int j = blockIdx.x * blockDim.x + threadIdx.x;
	//int i = blockIdx.y * blockDim.y + threadIdx.y;
	 double f[] = {0, -1, 0, -1, 5, -1, 0, -1, 0};
	//float f[] = {0, 0, 0, 0, 1, 0, 0, 0, 0};
	// double f[] = {-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,-0.4,};
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int maskRowsRadius = FILTER_SIZE / 2;
	int maskColsRadius = FILTER_SIZE / 2;
	float accum;

	for (int k = 0; k < channels; k++) {
		if (row < height && col < width) {
			accum = 0;
			int startRow = row - maskRowsRadius;
			int startCol = col - maskColsRadius;

			for (int i = 0; i < FILTER_SIZE; i++) {

				for (int j = 0; j < FILTER_SIZE; j++) {

					int currentRow = startRow + i;
					int currentCol = startCol + j;

					if (currentRow >= 0 && currentRow < height && currentCol >= 0 && currentCol < width) {

						accum += pixelMap[(currentRow * width + currentCol) * channels + k] *
							f[i * FILTER_SIZE + j];
					}
					else accum = 0;
				}

			}
			resultMap[(row * width + col) * channels + k] = (byte_t)((int)accum);
		}

	}
}

double coef[2] = { 1, 1.0 };
int filters[2][3][3] = {
	{
		{1,1,1},
		{1,1,1},
		{1,1,1}
	},
	{
		{0,-1,0},
		{-1,5,-1},
		{0,-1,0}
	}
};

int main(char** argv, int argc) {
	byte_t* pixels = NULL;
	byte_t* d_pixelMap, * d_resultMap, * h_resultMap;
	imgsize_t width = 0, height = 0, size = 0;
	auto inputImage = importPPM("lena.ppm");
	auto outputImage = Image_new(inputImage->width, inputImage->height, inputImage->channels);

	pixels = inputImage->data;
	h_resultMap = outputImage->data;
	size = width * height * inputImage->channels;
	//int* flatFilter = (int*)flattenArray((void**)filters[BoxBlur], 3, 3, sizeof(int));
	int flatFilter[] = { 0,-1,0,-1,5,-1,0,-1,0 };
	// byte_t* d_pixelMap, *d_resultMap, *h_resultMap;
	int* d_filter;
	gpuErrchk(hipMalloc((void**)&d_filter, sizeof(int) * 3 * 3));
	gpuErrchk(hipMalloc((void**)&d_pixelMap, sizeof(byte_t) * size));
	gpuErrchk(hipMalloc((void**)&d_resultMap, sizeof(byte_t) * size));
	//---cpy

	gpuErrchk(hipMemcpy(d_pixelMap, pixels, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_filter, flatFilter, sizeof(int) * 9, hipMemcpyHostToDevice));
	//DO STUFF
	/*
	Declare and allocate host and device memory. <
	Initialize host data. <
	Transfer data from the host to the device. <
	Execute one or more kernels. <
	Transfer results from the device to the host. <
	*/
	dim3 numberOfBlocks(ceil(width / 32), ceil(height / 32));
	dim3 threadsPerBlock(32, 32);
	convolution (d_pixelMap, d_filter, coef[BoxBlur], d_resultMap, width, height, inputImage->channels);

	gpuErrchk(hipPeekAtLastError());
	h_resultMap = (byte_t*)malloc(sizeof(byte_t) * size);

	gpuErrchk(hipMemcpy(h_resultMap, d_resultMap, size, hipMemcpyDeviceToHost));

	// hipDeviceSynchronize();
	// loadbmp_encode_file("lena2.bmp", h_resultMap, width, height, LOADBMP_RGB);
	outputImage->data = h_resultMap;
	exportPPM("output.ppm", outputImage);
	
	free(pixels);
	free(h_resultMap);
	// free(flatFilter);
	// fclose(fp);
	hipFree(d_filter);
	hipFree(d_resultMap);
	hipFree(d_pixelMap);
	return 0;
}